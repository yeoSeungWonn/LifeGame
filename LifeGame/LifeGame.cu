#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define WIDTH 512
#define HEIGHT 512

#define DEAD '-'
#define LIVE '*'
#define NONE '+'

int main(int argc, char** argv) {
	if (argc < 2) {
		perror("parameter more than 1");
		exit(1);
	}

	char width = WIDTH;
	char height = HEIGHT;
	int turnLimit = 500;
	int turn = 0;

	char* gamefieldParallelHost;

	srand((unsigned)time(NULL));
	
	for (int i = 0; i < width * height; i++) {
		gamefieldParallelHost[i] = rand() % 2;
	}

	for (int i = 0; i < width; i++) {
		gamefieldParallelHost[i] = NONE;
		gamefieldParallelHost[i + width * (height - 1)] = NONE;
	}

	for (int i = 0; i < height; i++) {
		gamefieldParallelHost[0 + width * i] = NONE; 
		gamefieldParallelHost[width - 1 + width * i] = NONE;
	}

	for (int i = 0; i < height; i++) {
		gamefieldParallelHost[0 + width * i] = NONE; 
		gamefieldParallelHost[width - 1 + width * i] = NONE; 
	}

	for (int i = 0; i < turnLimit; i++) {

	}
       
	return 0;
}