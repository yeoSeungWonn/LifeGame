#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "DS_timer.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <windows.h>


#define ROW 256
#define COL 256

#define NONE -1
#define DEAD 0
#define LIVE 1


void initfield(int* _gamefield1, int* _gamefield2);
void draw(int* _gamefield);
int checkMatrix(int* _gamefield1, int* _gamefield2);

__global__ void game(int* gamefieldOriginal, int* gamefieldBuffer)
{
	int width = blockDim.x;
	int height = gridDim.x;
	int blockID = threadIdx.x;

	int gridID = blockDim.x * blockIdx.x + blockID;
	

	if (gamefieldOriginal[gridID] == NONE) {
		gamefieldBuffer[gridID] = NONE;
	}
	else {
		int neighbors = 0;
		if (gamefieldOriginal[gridID - width - 1] == LIVE) { // upper left.
			neighbors++;
		}
		if (gamefieldOriginal[gridID - width] == LIVE) { // upper.
			neighbors++;
		}
		if (gamefieldOriginal[gridID - width + 1] == LIVE) { // upper right.
			neighbors++;
		}
		if (gamefieldOriginal[gridID - 1] == LIVE) { // left.
			neighbors++;
		}
		if (gamefieldOriginal[gridID + 1] == LIVE) { // right.
			neighbors++;
		}
		if (gamefieldOriginal[gridID + width - 1] == LIVE) { // lower left.
			neighbors++;
		}
		if (gamefieldOriginal[gridID + width] == LIVE) { // lower.
			neighbors++;
		}
		if (gamefieldOriginal[gridID + width + 1] == LIVE) { // lower right.
			neighbors++;
		}

		if (gamefieldOriginal[gridID] == DEAD) {
			if (neighbors == 3) {
				gamefieldBuffer[gridID] = LIVE;
			}
		}
		else if (gamefieldOriginal[gridID] == LIVE) {
			if (neighbors < 2 || neighbors > 3) {
				gamefieldBuffer[gridID] = DEAD;
			}
		}
	}
}

__global__ void copy(int* gamefieldOriginal, int* gamefieldBuffer) {
	int width = blockDim.x;
	int height = gridDim.x;
	int blockID = threadIdx.x;

	int gridID = blockDim.x * blockIdx.x + blockID;
	gamefieldOriginal[gridID] = gamefieldBuffer[gridID];

}

int main()
{
	DS_timer timer(2);
	timer.setTimerName(0, "CUDA Total");
	timer.setTimerName(1, "CPU Total");
	timer.initTimers();

	srand(time(NULL));
	int width = COL;
	int height = ROW;

	int size = sizeof(int) * width * height;

	int* gamefield;
	int term = 500;
	int count = 0;

	printf("%d * %d, %d games", ROW, COL, term);

	int* gamefieldParallelHost;
	int* gamefieldParallelCUDA;
	int* gamefieldBufferCUDA;
	int* gamefieldSerialHost;
	int* gamefieldBufferHost;

	hipMalloc(&gamefieldParallelCUDA, size);
	hipMalloc(&gamefieldBufferCUDA, size);

	gamefieldParallelHost = new int[width * height];
	gamefieldSerialHost = new int[width * height];
	gamefieldBufferHost = new int[width * height];

	memset(gamefieldParallelHost, 0, size);
	memset(gamefieldSerialHost, 0, size);
	memset(gamefieldBufferHost, 0, size);

	initfield(gamefieldParallelHost, gamefieldSerialHost);

	dim3 dimBlock(width);
	dim3 dimGrid(height);

	timer.onTimer(0);
	hipMemcpy(gamefieldBufferCUDA, gamefieldParallelHost, size, hipMemcpyHostToDevice);
	hipMemcpy(gamefieldParallelCUDA, gamefieldParallelHost, size, hipMemcpyHostToDevice);

	while (count < term)
	{

		game << <dimGrid, dimBlock >> > (gamefieldParallelCUDA, gamefieldBufferCUDA);
		copy << <dimGrid, dimBlock >> > (gamefieldParallelCUDA, gamefieldBufferCUDA);
		count++;
	}
	hipDeviceSynchronize();
	hipMemcpy(gamefieldParallelHost, gamefieldParallelCUDA, size, hipMemcpyDeviceToHost);
	timer.offTimer(0);

	count = 0;
	timer.onTimer(1);
	memcpy(gamefieldBufferHost, gamefieldSerialHost, size);

	while (count < term) {
		for (int i = 0; i < ROW * COL; i++) {
			if (gamefieldSerialHost[i] == NONE) {
				gamefieldBufferHost[i] = NONE;
			}
			else {
				int neighbors = 0;
				if (gamefieldSerialHost[i - width - 1] == LIVE) { // upper left.
					neighbors++;
				}
				if (gamefieldSerialHost[i - width] == LIVE) { // upper.
					neighbors++;
				}
				if (gamefieldSerialHost[i - width + 1] == LIVE) { // upper right.
					neighbors++;
				}
				if (gamefieldSerialHost[i - 1] == LIVE) { // left.
					neighbors++;
				}
				if (gamefieldSerialHost[i + 1] == LIVE) { // right.
					neighbors++;
				}
				if (gamefieldSerialHost[i + width - 1] == LIVE) { // lower left.
					neighbors++;
				}
				if (gamefieldSerialHost[i + width] == LIVE) { // lower.
					neighbors++;
				}
				if (gamefieldSerialHost[i + width + 1] == LIVE) { // lower right.
					neighbors++;
				}

				if (gamefieldSerialHost[i] == DEAD) {
					if (neighbors == 3) {
						gamefieldBufferHost[i] = LIVE;
					}
				}
				else if (gamefieldSerialHost[i] == LIVE) {
					if (neighbors < 2 || neighbors > 3) {
						gamefieldBufferHost[i] = DEAD;
					}
				}
			}
			
		}
		memcpy(gamefieldSerialHost, gamefieldBufferHost, size);
		count++;
	}
	timer.offTimer(1);
	timer.printTimer();

	if (checkMatrix(gamefieldParallelHost, gamefieldSerialHost)) {
		printf("같다\n");
	}
	else
		printf("다르다");

	hipFree(gamefieldParallelCUDA);
	hipFree(gamefieldBufferCUDA);
	
	delete[] gamefieldParallelHost; delete[] gamefieldSerialHost; delete[] gamefieldBufferHost;

	return 0;
}

void initfield(int* _gamefield1, int* _gamefield2)
{
	for (int i = 0; i < ROW * COL; i++)
		_gamefield1[i] = rand() % 2;

	for (int i = 0; i < COL; i++)
	{
		_gamefield1[i] = NONE; // 맨 위
		_gamefield1[i + COL * (ROW - 1)] = NONE; // 맨 아래
	}

	for (int i = 0; i < ROW; i++)
	{
		_gamefield1[COL * i] = NONE; // 맨 왼쪽
		_gamefield1[COL * (i + 1) - 1] = NONE; // 맨 오른쪽
	}

	for (int i = 0; i < ROW * COL; i++) {
		_gamefield2[i] = _gamefield1[i];
	}
}

void draw(int* _gamefield)
{
	for (int i = 0; i < ROW; i++)
	{
		for (int j = 0; j < COL; j++)
		{
			printf("[%2d]", _gamefield[i * ROW + j]);
		}
		printf("\n");
	}
}


int checkMatrix(int* _gamefield1, int* _gamefield2) {
	for (int i = 0; i < ROW * COL; i++) {
		if (_gamefield1[i] != _gamefield2[i]) {
			return 0;
		}
	}
	return 1;
}
